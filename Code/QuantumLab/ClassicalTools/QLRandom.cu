#include "hip/hip_runtime.h"
//=============================================================================
// FILENAME : QLRandom.cpp
// 
// DESCRIPTION:
//
//
// REVISION: [dd/mm/yy]
//  [12/09/2022 nbale]
//=============================================================================
#include "QuantumLabPCH.h"

__BEGIN_NAMESPACE

__global__ void _QL_LAUNCH_BOUND
_kernalAllocateSeedTable(UINT* pDevicePtr, UINT uiSeed)
{
    UINT uiThread = threadIdx.x;
    QLRandom::_deviceAsignSeeds(pDevicePtr, uiSeed + uiThread, uiThread);
}

__global__ void _QL_LAUNCH_BOUND
_kernalInitialXORWOW(hiprandState* states, UINT uiSeed)
{
    UINT uiThread = threadIdx.x;
    hiprand_init(uiSeed, uiThread, 0, &states[uiThread]);
}

__global__ void _QL_LAUNCH_BOUND
_kernalInitialPhilox(hiprandStatePhilox4_32_10_t* states, UINT uiSeed)
{
    UINT uiThread = threadIdx.x;
    hiprand_init(uiSeed, uiThread, 0, &states[uiThread]);
}

__global__ void _QL_LAUNCH_BOUND
_kernalInitialMRG(hiprandStateMRG32k3a* states, UINT uiSeed)
{
    UINT uiThread = threadIdx.x;
    hiprand_init(uiSeed, uiThread, 0, &states[uiThread]);
}

__global__ void _QL_LAUNCH_BOUND
_kernalInitialSobel32(hiprandStateSobol32* states, hiprandDirectionVectors32_t* dirs, UINT uiSeed)
{
    UINT uiThread = threadIdx.x;
    hiprand_init(dirs[uiThread], uiSeed % 16, &states[uiThread]);
}

__global__ void _QL_LAUNCH_BOUND
_kernalInitialScrambledSobel32(hiprandStateScrambledSobol32* states, UINT* consts, hiprandDirectionVectors32_t* dirs, UINT uiSeed)
{
    UINT uiThread = threadIdx.x;
    hiprand_init(dirs[uiThread], consts[uiThread], uiSeed % __SOBEL_OFFSET_MAX, &states[uiThread]);
}

QLRandom::~QLRandom()
{

    switch (m_eRandomType)
    {
    case ERandom::ER_Schrage:
    {
        checkCudaErrors(hipFree(m_pDeviceSeedTable));
    }
    break;
    case ERandom::ER_MRG32K3A:
    {
        checkCudaErrors(hiprandDestroyGenerator(m_HGen));
        checkCudaErrors(hipFree(m_deviceBuffer));
        checkCudaErrors(hipFree(m_pDeviceRandStatesMRG));
    }
    break;
    case ERandom::ER_PHILOX4_32_10:
    {
        checkCudaErrors(hiprandDestroyGenerator(m_HGen));
        checkCudaErrors(hipFree(m_deviceBuffer));
        checkCudaErrors(hipFree(m_pDeviceRandStatesPhilox));
    }
    break;
    case ERandom::ER_QUASI_SOBOL32:
    {
        checkCudaErrors(hiprandDestroyGenerator(m_HGen));
        checkCudaErrors(hipFree(m_deviceBuffer));
        checkCudaErrors(hipFree(m_pDeviceRandStatesSobol32));
        checkCudaErrors(hipFree(m_pDeviceSobolDirVec));
    }
    break;
    case ERandom::ER_SCRAMBLED_SOBOL32:
    {
        checkCudaErrors(hiprandDestroyGenerator(m_HGen));
        checkCudaErrors(hipFree(m_deviceBuffer));
        checkCudaErrors(hipFree(m_pDeviceRandStatesScrambledSobol32));
        checkCudaErrors(hipFree(m_pDeviceSobolDirVec));
        checkCudaErrors(hipFree(m_pDeviceSobelConsts));
    }
    break;
    case ERandom::ER_XORWOW:
    default:
    {
        checkCudaErrors(hiprandDestroyGenerator(m_HGen));
        checkCudaErrors(hipFree(m_deviceBuffer));
        checkCudaErrors(hipFree(m_pDeviceRandStatesXORWOW));
    }
    break;
    }
}

//Initial XORWOW only support 512 threads per block
void QLRandom::InitialStatesXORWOW()
{
    checkCudaErrors(hipMalloc((void**)&m_pDeviceRandStatesXORWOW, sizeof(hiprandState) * m_uiMaxThread));
    _kernalInitialXORWOW << <1, m_uiMaxThread >> > (m_pDeviceRandStatesXORWOW, m_uiHostSeed);
}

//Initial Philox only support 256 threads per block
void QLRandom::InitialStatesPhilox()
{
    checkCudaErrors(hipMalloc((void**)&m_pDeviceRandStatesPhilox, sizeof(hiprandStatePhilox4_32_10_t) * m_uiMaxThread));
    _kernalInitialPhilox << <1, m_uiMaxThread >> > (m_pDeviceRandStatesPhilox, m_uiHostSeed);
}

//Initial MRG only support 256 threads per block
void QLRandom::InitialStatesMRG()
{
    checkCudaErrors(hipMalloc((void**)&m_pDeviceRandStatesMRG, sizeof(hiprandStateMRG32k3a) * m_uiMaxThread));
    _kernalInitialMRG << <1, m_uiMaxThread >> > (m_pDeviceRandStatesMRG, m_uiHostSeed);
}

void QLRandom::InitialStatesSobol32()
{
    //support only 20000 dimensions, so using _HC_Volumn instead
    checkCudaErrors(hipMalloc((void**)&m_pDeviceRandStatesSobol32, sizeof(hiprandStateSobol32) * m_uiMaxThread));
    checkCudaErrors(hipMalloc((void**)&m_pDeviceSobolDirVec, sizeof(hiprandDirectionVectors32_t) * m_uiMaxThread));

    //int[32]
    hiprandDirectionVectors32_t* hostVectors32;
    checkCudaErrors(hiprandGetDirectionVectors32(&hostVectors32, HIPRAND_DIRECTION_VECTORS_32_JOEKUO6));
    checkCudaErrors(hipMemcpy(m_pDeviceSobolDirVec, hostVectors32, sizeof(hiprandDirectionVectors32_t) * m_uiMaxThread, hipMemcpyHostToDevice));

    _kernalInitialSobel32 << <1, m_uiMaxThread >> > (m_pDeviceRandStatesSobol32, m_pDeviceSobolDirVec, m_uiHostSeed);
}

void QLRandom::InitialStatesScrambledSobol32()
{
    checkCudaErrors(hipMalloc((void**)&m_pDeviceRandStatesScrambledSobol32, sizeof(hiprandStateScrambledSobol32) * m_uiMaxThread));
    checkCudaErrors(hipMalloc((void**)&m_pDeviceSobolDirVec, sizeof(hiprandDirectionVectors32_t) * m_uiMaxThread));
    checkCudaErrors(hipMalloc((void**)&m_pDeviceSobelConsts, sizeof(UINT) * m_uiMaxThread));

    hiprandDirectionVectors32_t* hostVectors32;
    checkCudaErrors(hiprandGetDirectionVectors32(&hostVectors32, HIPRAND_SCRAMBLED_DIRECTION_VECTORS_32_JOEKUO6));
    checkCudaErrors(hipMemcpy(m_pDeviceSobolDirVec, hostVectors32, sizeof(hiprandDirectionVectors32_t) * m_uiMaxThread, hipMemcpyHostToDevice));

    UINT* hostScrambleConstants32;
    checkCudaErrors(hiprandGetScrambleConstants32(&hostScrambleConstants32));
    checkCudaErrors(hipMemcpy(m_pDeviceSobelConsts, hostScrambleConstants32, sizeof(UINT) * m_uiMaxThread, hipMemcpyHostToDevice));

    _kernalInitialScrambledSobel32 << <1, m_uiMaxThread >> > (m_pDeviceRandStatesScrambledSobol32, m_pDeviceSobelConsts, m_pDeviceSobolDirVec, m_uiHostSeed);
}

void QLRandom::InitialTableSchrage()
{
    checkCudaErrors(hipMalloc((void**)&m_pDeviceSeedTable, sizeof(UINT) * m_uiMaxThread));
    _kernalAllocateSeedTable << <1, m_uiMaxThread >> > (m_pDeviceSeedTable, m_uiMaxThread);
}

//to do add options to initialize the random
QLRandomInitializer::QLRandomInitializer(ERandom eRandom, UINT uiSeed)
    : m_pRandom(NULL)
    , m_pDeviceRandom(NULL)
{
    if (0 == uiSeed)
    {
        uiSeed = appGetTimeStamp();
    }

    m_pRandom = new QLRandom(uiSeed, _QL_LAUNCH_MAX_THREAD, eRandom);
    checkCudaErrors(hipMalloc((void**)&m_pDeviceRandom, sizeof(QLRandom)));
    checkCudaErrors(hipMemcpy(m_pDeviceRandom, m_pRandom, sizeof(QLRandom), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(__r), &m_pDeviceRandom, sizeof(QLRandom*)));

    printf("random initialed\n");

    _hostRandomPointer = m_pRandom;
}

QLRandomInitializer::~QLRandomInitializer()
{
    checkCudaErrors(hipFree(m_pDeviceRandom));
    appSafeDelete(m_pRandom);
}

__constant__ QLRandom* __r;
//QLRandomInitializer GRandom;
QLRandom* _hostRandomPointer = NULL;

__END_NAMESPACE

//=============================================================================
// END OF FILE
//=============================================================================
