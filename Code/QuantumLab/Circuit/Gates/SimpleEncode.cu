#include "hip/hip_runtime.h"
//=============================================================================
// FILENAME : SimpleEncode.cu
// 
// DESCRIPTION:
// 
//
// REVISION: [dd/mm/yy]
//  [06/05/2022 nbale]
//=============================================================================

#include "QuantumLabPCH.h"

__BEGIN_NAMESPACE

#pragma region kernels

/**
* 
*/
__global__ void _QL_LAUNCH_BOUND
_kernelSE_FetchDegrees(Real* YLst, Real* ZLst, const QLComplex * __restrict__ vBuffer, UINT vectorCount, UINT lengthCount)
{
    const UINT idx = (threadIdx.x + blockIdx.x * blockDim.x);

    if (idx < vectorCount * lengthCount)
    {
        UINT uiV = idx / lengthCount;
        UINT uiD = idx % lengthCount;

        YLst[uiD * vectorCount + uiV] = vBuffer[uiV * lengthCount + uiD].x * PI;
        ZLst[uiD * vectorCount + uiV] = -vBuffer[uiV * lengthCount + uiD].y * PI;
    }
}

#pragma endregion

/**
* ry rz CNOT
*/
QLGate QLAPI SimpleEncodeOneVector(const QLComplex* hostv, BYTE qubits, UINT uiVLength)
{
    QLGate ret;
    ret.AddQubits(qubits);

    QLGate cnot(EBasicOperation::EBO_CX);

    BYTE toaddbyte = 0;
    UBOOL bLastLevelCnotAdded = FALSE;

    for (UINT i = 0; i < uiVLength; ++i)
    {
        QLGate ry(EBasicOperation::EBO_RY, hostv[i].x * PI);
        QLGate rz(EBasicOperation::EBO_RZ, hostv[i].y * PI);

        ret.AppendGate(ry, toaddbyte);
        ret.AppendGate(rz, toaddbyte);

        toaddbyte = toaddbyte + 1;
        if (toaddbyte == qubits)
        {
            for (BYTE j = 0; j < qubits; ++j)
            {
                if (0 == j)
                {
                    ret.AppendGate(cnot, j, qubits - 1);
                }
                else
                {
                    ret.AppendGate(cnot, j, j - 1);
                }
            }
            toaddbyte = 0;
            bLastLevelCnotAdded = TRUE;
        }
        else
        {
            bLastLevelCnotAdded = FALSE;
        }
    }

    if (!bLastLevelCnotAdded)
    {
        for (BYTE j = 0; j < qubits; ++j)
        {
            if (0 == j)
            {
                ret.AppendGate(cnot, j, qubits - 1);
            }
            else
            {
                ret.AppendGate(cnot, j, j - 1);
            }
        }
    }

    return ret;
}

QLGate QLAPI SimpleEncodeVectors(const QLComplex* hostv, BYTE vectorCountPower, BYTE qubits, UINT uiVLength)
{
    QLGate ret;
    //const BYTE uiVPower = static_cast<BYTE>(MostSignificantPowerTwo(uiVLength));
    ret.AddQubits(vectorCountPower + qubits);
    const UINT uiVectorCount = static_cast<UINT>(1UL << vectorCountPower);
    const UINT uiNumberCount = uiVectorCount * uiVLength;

    QLComplex* devicev = NULL;
    Real* devicedegreeY = NULL;
    Real* devicedegreeZ = NULL;
    checkCudaErrors(hipMalloc((void**)&devicev, sizeof(QLComplex) * uiNumberCount));
    checkCudaErrors(hipMalloc((void**)&devicedegreeY, sizeof(Real) * uiNumberCount));
    checkCudaErrors(hipMalloc((void**)&devicedegreeZ, sizeof(Real) * uiNumberCount));
    checkCudaErrors(hipMemcpy(devicev, hostv, sizeof(QLComplex) * uiNumberCount, hipMemcpyHostToDevice));
    UINT uiBlock = Ceil(uiNumberCount, _QL_LAUNCH_MAX_THREAD);
    _kernelSE_FetchDegrees << <uiBlock, _QL_LAUNCH_MAX_THREAD >> > (devicedegreeY, devicedegreeZ, devicev, uiVectorCount, uiVLength);

    Real* hostdegreeY = (Real*)malloc(sizeof(Real) * uiNumberCount);
    Real* hostdegreeZ = (Real*)malloc(sizeof(Real) * uiNumberCount);
    checkCudaErrors(hipMemcpy(hostdegreeY, devicedegreeY, sizeof(Real) * uiNumberCount, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(hostdegreeZ, devicedegreeZ, sizeof(Real) * uiNumberCount, hipMemcpyDeviceToHost));
    checkCudaErrors(hipFree(devicev));
    checkCudaErrors(hipFree(devicedegreeY));
    checkCudaErrors(hipFree(devicedegreeZ));

    //QLGate fryz = FRyz(hostY + idx, hostZ + idx, static_cast<UINT>(bits.Num()));
    QLGate cnot(EBasicOperation::EBO_CX);
    QLGate h(EBasicOperation::EBO_H);

    BYTE toaddbyte = 0;
    UBOOL bLastLevelCnotAdded = FALSE;
    TArray<BYTE> bits;
    for (BYTE b = 0; b < vectorCountPower; ++b)
    {
        bits.AddItem(vectorCountPower + qubits - 1 - b);
        ret.AppendGate(h, vectorCountPower + qubits - 1 - b);
    }
    bits.AddItem(0);

    for (UINT i = 0; i < uiVLength; ++i)
    {
        QLGate fryz = FRyz(hostdegreeY + i * uiVectorCount, hostdegreeZ + i * uiVectorCount, static_cast<UINT>(vectorCountPower + 1));
        //QLGate fry = FRy(hostdegreeY + i * uiVectorCount, static_cast<UINT>(vectorCountPower + 1));

        TArray<Real> degrees;
        for (UINT k = 0; k < uiVectorCount; ++k)
        {
            degrees.AddItem(hostdegreeY[i * uiVectorCount + k]);
            degrees.AddItem(hostdegreeZ[i * uiVectorCount + k]);
        }
        appGeneral(_T("degress: %s\n"), appToString(degrees));

        //For example, if we have 16 vectors (length-8 complex), we need
        //6543-0, 6543-1, 6543-2, ...
        
        bits[vectorCountPower] = toaddbyte;
        ret.AppendGate(fryz, bits);
        //ret.AppendGate(fry, bits);

        toaddbyte = toaddbyte + 1;
        if (toaddbyte == qubits)
        {
            for (BYTE j = 0; j < qubits; ++j)
            {
                if (0 == j)
                {
                    ret.AppendGate(cnot, j, qubits - 1);
                }
                else
                {
                    ret.AppendGate(cnot, j, j - 1);
                }
            }
            toaddbyte = 0;
            bLastLevelCnotAdded = TRUE;
        }
        else
        {
            bLastLevelCnotAdded = FALSE;
        }
    }

    if (!bLastLevelCnotAdded)
    {
        for (BYTE j = 0; j < qubits; ++j)
        {
            if (0 == j)
            {
                ret.AppendGate(cnot, j, qubits - 1);
            }
            else
            {
                ret.AppendGate(cnot, j, j - 1);
            }
        }
    }

    return ret;
}

__END_NAMESPACE


//=============================================================================
// END OF FILE
//=============================================================================